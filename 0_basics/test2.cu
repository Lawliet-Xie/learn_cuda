// index data
// z[i] = x[i] + y[i]
// for loop
// thread z[i]
// memory allocation
// memory copy  gpu_mem != cpu_mem
// kernel func
// memory copy


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>



__global__ void vecAdd(const double *x, const double *y, double *z, int count)
{
    const int index = blockDim.x * blockIdx.x + threadIdx.x;
    // t00 t01 t02 t10 t11 t12 t20 t21 t22
    // t21: blockDim.x = 3, blockIdx.x = 2, threadIdx.x = 1 ==> index = 7
    if (index < count)
    {
        z[index] = x[index] + y[index];
    }
}


void vecAdd_cpu(const double *x, const double *y, double *z, int count)
{
    for (int i = 0; i < count; ++i)
    {
        z[i] = x[i] + y[i];
    }
}





// x[] + y[] = z[]
int main()
{
    const int N = 1000;
    const int M = sizeof(double) * N;

    // cpu mem alloc
    double *h_x = (double *) malloc(M);  // host
    double *h_y = (double *) malloc(M);
    double *h_z = (double *) malloc(M);
    double *cpu_res = (double *) malloc(M);

    // init
    for (int i = 0; i < N; ++i)
    {
        h_x[i] = 1;
        h_y[i] = 2;
    }

    // gpu mem alloc
    double *d_x, *d_y, *d_z;
    hipMalloc((void **) &d_x, M);
    hipMalloc((void **) &d_y, M);
    hipMalloc((void **) &d_z, M);

    // cpu to gpu
    hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, M, hipMemcpyHostToDevice);

    const int block_size = 128;
    const int grid_size = (N + block_size - 1) / block_size;

    // kernel func
    vecAdd<<<grid_size, block_size>>>(d_x, d_y, d_z, N);

    vecAdd_cpu(h_x, h_y, cpu_res, N);

    // gpu to cpu
    hipMemcpy(h_z, d_z, M, hipMemcpyDeviceToHost);

    bool error = false;
    for (int i = 0; i < N; ++i)
    {
        if (fabs(cpu_res[i] - h_z[i]) > (1.0e-10))
        {
            error = true;
        }
    }
    printf("Result: %s\n", error?"Error":"Pass");

    free(h_x);
    free(h_y);
    free(h_z);
    free(cpu_res);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);


    return 0;
}



